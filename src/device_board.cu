#include "device_board.h"

Board newDeviceBoard() {
	Board d_board;
	hipMalloc(&d_board, getBoardSize() * sizeof(char));
	return d_board;
}

void freeDeviceBoard(Board board) {
	hipFree(board);
}

void copyBoardToDevice(Board board, Board d_board) {
	hipMemcpy(d_board, board, getBoardSize() * sizeof(char), hipMemcpyHostToDevice);
}

void copyDeviceToBoard(Board d_board, Board board) {
	hipMemcpy(board, d_board, getBoardSize() * sizeof(char), hipMemcpyDeviceToHost);
}

#define d_wrapCoord(c) ((c + d_getBoardDim()) % d_getBoardDim())

__device__ CellState d_getCellState(Board board, int x, int y) {
	return (board[d_getBoardDim() * d_wrapCoord(y) + d_wrapCoord(x)] == 1) ? ALIVE : DEAD;
}

__device__ void d_setCellState(Board board, int x, int y, CellState state) {
	board[d_getBoardDim() * d_wrapCoord(y) + d_wrapCoord(x)] = (state == ALIVE) ? 1 : 0;
}
