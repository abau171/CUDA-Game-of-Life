#include "hip/hip_runtime.h"
extern "C" {

#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include <gpu_gameoflife.h>

#define CUDA(X) X
//printf("CUDA: %s\n", hipGetErrorString(X))

#define WIDTH 32
#define HEIGHT 18
#define NUM_CELLS (WIDTH * HEIGHT)

static char board[WIDTH * HEIGHT];

void gpu_set_cell(int x, int y) {
    board[x + WIDTH * y] = 1;
}

void gpu_randomize_board(void) {
    srand(172);
    for (int y = 0; y < HEIGHT; y++) {
        for (int x = 0; x < WIDTH; x++) {
            if (rand() % 2 == 0) {
                gpu_set_cell(x, y);
            }
        }
    }
}

__device__
static inline char wrap_x(int x) {
    return (x + WIDTH) % WIDTH;
}

__device__
static inline char wrap_y(int y) {
    return (y + HEIGHT) % HEIGHT;
}

__global__
static void d_next_gen(char* board, char* buffer_board) {
    int cell_index = blockIdx.x * blockDim.x + threadIdx.x;
    if (cell_index >= NUM_CELLS) return;
    int y = cell_index / WIDTH;
    int x = cell_index % WIDTH;
    char alive = board[cell_index];
    int num_neighbors = 0;
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            if (dx == 0 && dy == 0) continue;
            num_neighbors += board[wrap_x(x + dx) + WIDTH * wrap_y(y + dy)];
        }
    }
    if (alive && (num_neighbors < 2 || num_neighbors > 3)) {
        buffer_board[cell_index] = 0;
    } else if (!alive && num_neighbors == 3) {
        buffer_board[cell_index] = 1;
    } else {
        buffer_board[cell_index] = alive;
    }
}

void gpu_next_n_gens(int n) {
    char* d_board;
    char* d_buffer_board;

    CUDA(hipMalloc(&d_board, NUM_CELLS * sizeof(char)));
    CUDA(hipMalloc(&d_buffer_board, NUM_CELLS * sizeof(char)));
    CUDA(hipMemcpy(d_board, board, NUM_CELLS * sizeof(char), hipMemcpyHostToDevice));

    for (int i = 0; i < n; i++)  {
        d_next_gen<<<(NUM_CELLS + 64 - 1) / 64, 64>>>(d_board, d_buffer_board);

        char* tmp = d_board;
        d_board = d_buffer_board;
        d_buffer_board = tmp;
    }

    CUDA(hipMemcpy(board, d_board, NUM_CELLS * sizeof(char), hipMemcpyDeviceToHost));
    CUDA(hipFree(d_board));
    CUDA(hipFree(d_buffer_board));
}

void gpu_print_board(void) {
    putchar('O');
    for (int i = 0; i < 2 * WIDTH - 1; i++) {
        putchar('-');
    }
    putchar('O');
    putchar('\n');
    for (int y = 0; y < HEIGHT; y++) {
        putchar('|');
        for (int x = 0; x < WIDTH; x++) {
            if (x > 0) {
                putchar(' ');
            }
            putchar(board[x + WIDTH * y] ? 'X' : ' ');
        }
        putchar('|');
        putchar('\n');
    }
    putchar('O');
    for (int i = 0; i < 2 * WIDTH - 1; i++) {
        putchar('-');
    }
    putchar('O');
    putchar('\n');
}

}

