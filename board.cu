#include <stdio.h>

#include "board.h"

Board newBoard() {
	Board board = (Board) malloc(BOARD_SIZE * sizeof(char));
	for (int i = 0; i < BOARD_SIZE; i++) {
		board[i] = 0;
	}
	return board;
}

void freeBoard(Board board) {
	free(board);
}

Board newDeviceBoard() {
	Board d_board;
	hipMalloc(&d_board, BOARD_SIZE * sizeof(char));
	return d_board;
}

void freeDeviceBoard(Board board) {
	hipFree(board);
}

void copyBoardToDevice(Board board, Board d_board) {
	hipMemcpy(d_board, board, BOARD_SIZE * sizeof(char), hipMemcpyHostToDevice);
}

void copyDeviceToBoard(Board d_board, Board board) {
	hipMemcpy(board, d_board, BOARD_SIZE * sizeof(char), hipMemcpyDeviceToHost);
}

#define wrapCoord(c) ((c + BOARD_DIM) % BOARD_DIM)

CellState getCellState(Board board, int x, int y) {
	return (board[BOARD_DIM * wrapCoord(y) + wrapCoord(x)] == 1) ? ALIVE : DEAD;
}

void setCellState(Board board, int x, int y, CellState state) {
	board[BOARD_DIM * wrapCoord(y) + wrapCoord(x)] = (state == ALIVE) ? 1 : 0;
}

static void printBoardHorizontalBorder() {
	printf("%c", CORNER_BORDER);
	for (int col = 0; col < 2 * BOARD_DIM + 1; col++) {
		printf("%c", HORIZONTAL_BORDER);
	}
	printf("%c\n", CORNER_BORDER);
}

void printBoard(Board board) {
	printBoardHorizontalBorder();
	for (int y = 0; y < BOARD_DIM; y++) {
		printf("%c", VERTICAL_BORDER);
		for (int x = 0; x < BOARD_DIM; x++) {
			printf(" %c", (getCellState(board, x, y) == ALIVE) ? '#' : ' ');
		}
		printf(" %c%c\n", VERTICAL_BORDER);
	}
	printBoardHorizontalBorder();
}
