#include "device_board.h"

Board newDeviceBoard() {
	Board d_board;
	hipMalloc(&d_board, BOARD_SIZE * sizeof(char));
	return d_board;
}

void freeDeviceBoard(Board board) {
	hipFree(board);
}

void copyBoardToDevice(Board board, Board d_board) {
	hipMemcpy(d_board, board, BOARD_SIZE * sizeof(char), hipMemcpyHostToDevice);
}

void copyDeviceToBoard(Board d_board, Board board) {
	hipMemcpy(board, d_board, BOARD_SIZE * sizeof(char), hipMemcpyDeviceToHost);
}
