#include "device_board.h"

Board newDeviceBoard() {
	Board d_board;
	hipMalloc(&d_board, BOARD_SIZE * sizeof(char));
	return d_board;
}

void freeDeviceBoard(Board board) {
	hipFree(board);
}

void copyBoardToDevice(Board board, Board d_board) {
	hipMemcpy(d_board, board, BOARD_SIZE * sizeof(char), hipMemcpyHostToDevice);
}

void copyDeviceToBoard(Board d_board, Board board) {
	hipMemcpy(board, d_board, BOARD_SIZE * sizeof(char), hipMemcpyDeviceToHost);
}

#define wrapCoord(c) ((c + BOARD_DIM) % BOARD_DIM)

__device__ CellState d_getCellState(Board board, int x, int y) {
	return (board[BOARD_DIM * wrapCoord(y) + wrapCoord(x)] == 1) ? ALIVE : DEAD;
}

__device__ void d_setCellState(Board board, int x, int y, CellState state) {
	board[BOARD_DIM * wrapCoord(y) + wrapCoord(x)] = (state == ALIVE) ? 1 : 0;
}
