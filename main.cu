
#include <hip/hip_runtime.h>
#include <stdio.h>

#define BOARD_DIM 1024
#define NUM_GENERATIONS 500

#define USE_GPU false

#define BOARD_CORNER_BORDER "O"
#define BOARD_HORIZONTAL_BORDER "--"
#define BOARD_VERTICAL_BORDER "|"

typedef char* Board;

__device__ int d_getWrappedIndex(int i) {
	// wraps to within board bounds between -BOARD_DIM and +infinity
	return (i + BOARD_DIM) % BOARD_DIM;
}

__global__ void d_computeNextBoard(Board d_prevBoard, Board d_nextBoard) {
	int cellIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int col = cellIndex % BOARD_DIM;
	int row = (cellIndex - col) / BOARD_DIM;
	int numLivingNeighbors = 0;
	for (int dRow = -1; dRow < 2; dRow++) {
		for (int dCol = -1; dCol < 2; dCol++) {
			if (dRow == 0 && dCol == 0) {
				continue;
			}
			int neighborCellIndex = BOARD_DIM * d_getWrappedIndex(row + dRow) + d_getWrappedIndex(col + dCol);
			if (d_prevBoard[neighborCellIndex] > 0) {
				numLivingNeighbors++;
			}
		}
	}
	bool curStateAlive = d_prevBoard[BOARD_DIM * row + col];
	bool nextStateAlive;
	if (curStateAlive) {
		nextStateAlive = (numLivingNeighbors >= 2 && numLivingNeighbors <= 3);
	} else {
		nextStateAlive = (numLivingNeighbors == 3);
	}
	d_nextBoard[cellIndex] = nextStateAlive ? 1 : 0;
}

void printBoardHorizontalBorder() {
	printf(BOARD_CORNER_BORDER);
	for (int col = 0; col < BOARD_DIM; col++) {
		printf(BOARD_HORIZONTAL_BORDER);
	}
	printf("%s\n", BOARD_CORNER_BORDER);
}

void printBoard(Board board) {
	printBoardHorizontalBorder();
	for (int row = 0; row < BOARD_DIM; row++) {
		printf(BOARD_VERTICAL_BORDER);
		for (int col = 0; col < BOARD_DIM; col++) {
			printf(" %c", board[BOARD_DIM * row + col] > 0 ? '#' : ' ');
		}
		printf("%s\n", BOARD_VERTICAL_BORDER);
	}
	printBoardHorizontalBorder();
}

void printGeneration(Board board, int generation) {
	printf("\nGeneration %d\n", generation);
	printBoard(board);
}

Board newBoard() {
	Board board = (char*) malloc(BOARD_DIM * BOARD_DIM);
	return board;
}

void buildGliderBoard(Board board) {
	for (int row = 0; row < BOARD_DIM; row++) {
		for (int col = 0; col < BOARD_DIM; col++) {
			int cellIndex = BOARD_DIM * row + col;
			board[cellIndex] = 0;
		}
	}
	board[BOARD_DIM * 0 + 0] = 1;
	board[BOARD_DIM * 1 + 1] = 1;
	board[BOARD_DIM * 1 + 2] = 1;
	board[BOARD_DIM * 2 + 0] = 1;
	board[BOARD_DIM * 2 + 1] = 1;
}

int getWrappedIndex(int i) {
	// wraps to within board bounds between -BOARD_DIM and +infinity
	return (i + BOARD_DIM) % BOARD_DIM;
}

void nextGenerationCPU(Board* currentBoard_p, Board* bufferBoard_p) {
	Board prevBoard = *currentBoard_p;
	Board nextBoard = *bufferBoard_p;
	for (int row = 0; row < BOARD_DIM; row++) {
		for (int col = 0; col < BOARD_DIM; col++) {
			int cellIndex = BOARD_DIM * row + col;
			int numLivingNeighbors = 0;
			for (int dRow = -1; dRow < 2; dRow++) {
				for (int dCol = -1; dCol < 2; dCol++) {
					if (dRow == 0 && dCol == 0) {
						continue;
					}
					int neighborCellIndex = BOARD_DIM * getWrappedIndex(row + dRow) + getWrappedIndex(col + dCol);
					if (prevBoard[neighborCellIndex] > 0) {
						numLivingNeighbors++;
					}
				}
			}
			bool curStateAlive = prevBoard[BOARD_DIM * row + col];
			bool nextStateAlive;
			if (curStateAlive) {
				nextStateAlive = (numLivingNeighbors >= 2 && numLivingNeighbors <= 3);
			} else {
				nextStateAlive = (numLivingNeighbors == 3);
			}
			nextBoard[cellIndex] = nextStateAlive ? 1 : 0;
		}
	}
	*currentBoard_p = nextBoard;
	*bufferBoard_p = prevBoard;
}

void nextNGenerationsCPU(Board* currentBoard_p, Board* bufferBoard_p, int numGenerations) {
	for (int i = 0; i < numGenerations; i++) {
		nextGenerationCPU(currentBoard_p, bufferBoard_p);
	}
}

void nextNGenerationsGPU(Board* currentBoard_p, int numGenerations) {
	Board currentBoard = *currentBoard_p;

	Board d_board1;
	Board d_board2;
	Board* d_currentBoard = &d_board1;
	Board* d_bufferBoard = &d_board2;
	hipMalloc((void**) d_currentBoard, BOARD_DIM * BOARD_DIM);
	hipMalloc((void**) d_bufferBoard, BOARD_DIM * BOARD_DIM);

	hipMemcpy(*d_currentBoard, currentBoard, BOARD_DIM * BOARD_DIM, hipMemcpyHostToDevice);

	for (int i = 0; i < numGenerations; i++) {
		d_computeNextBoard<<<BOARD_DIM, BOARD_DIM>>>(*d_currentBoard, *d_bufferBoard);
		Board tmp = *d_currentBoard;
		*d_currentBoard = *d_bufferBoard;
		*d_bufferBoard = tmp;
	}

	hipMemcpy(currentBoard, *d_currentBoard, BOARD_DIM * BOARD_DIM, hipMemcpyDeviceToHost);

	hipFree(d_board1);
	hipFree(d_board2);

    hipDeviceReset();
}

int main() {
	Board board1 = newBoard();
	Board board2 = newBoard();
	Board* currentBoard_p = &board1;
	Board* bufferBoard_p = &board2;
	buildGliderBoard(*currentBoard_p);
	printGeneration(*currentBoard_p, 0);
	if (USE_GPU) {
		nextNGenerationsGPU(currentBoard_p, NUM_GENERATIONS);
	} else {
		nextNGenerationsCPU(currentBoard_p, bufferBoard_p, NUM_GENERATIONS);
	}
	printGeneration(*currentBoard_p, NUM_GENERATIONS);
	free(board1);
	free(board2);
	return 0;
}
