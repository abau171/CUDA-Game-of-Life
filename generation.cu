#include "hip/hip_runtime.h"
#include <stdio.h>

#include "generation.h"
#include "board.h"
#include "device_board.h"

static __device__ int d_getNumLivingNeighbors(Board board, int x, int y) {
	int numLivingNeighbors = 0;
	for (int dy = -1; dy <= 1; dy++) {
		for (int dx = -1; dx <= 1; dx++) {
			if (dx == 0 && dy == 0) {
				continue;
			}
			if (d_getCellState(board, x + dx, y + dy) == ALIVE) {
				numLivingNeighbors++;
			}
		}
	}
	return numLivingNeighbors;
}

static __device__ CellState d_getNextState(bool cellAlive, int numLivingNeighbors) {
	CellState nextState = DEAD;
	if (cellAlive) {
		if (numLivingNeighbors >= 2 && numLivingNeighbors <= 3) {
			nextState = ALIVE;
		}
	} else if (numLivingNeighbors == 3) {
		nextState = ALIVE;
	}
	return nextState;
}

static __device__ void d_nextGen(Board prevBoard, Board nextBoard) {
	bool cellAlive = (d_getCellState(prevBoard, threadIdx.x, threadIdx.y) == ALIVE);
	int numLivingNeighbors = d_getNumLivingNeighbors(prevBoard, threadIdx.x, threadIdx.y);
	CellState nextState = d_getNextState(cellAlive, numLivingNeighbors);
	d_setCellState(nextBoard, threadIdx.x, threadIdx.y, nextState);
}

static __device__ void d_swapBoards(Board** board1_pp, Board** board2_pp) {
	Board* tmp = *board1_pp;
	*board1_pp = *board2_pp;
	*board2_pp = tmp;
}

__global__ void d_nextNGens(Board d_board1, Board d_board2, int numGens) {
	Board* curBoard_p = &d_board1;
	Board* nextBoard_p = &d_board2;
	for (int i = 0; i < numGens; i++) {
		d_nextGen(*curBoard_p, *nextBoard_p);
		d_swapBoards(&curBoard_p, &nextBoard_p);
		__syncthreads();
	}
}

static Board* pickFinalBoard(Board* d_board1_p, Board* d_board2_p, int numGens) {
	if (numGens % 2 == 0) {
		return d_board1_p;
	} else {
		return d_board2_p;
	}
}

void nextNGens(Board origBoard, int numGens) {
	Board d_board1 = newDeviceBoard();
	Board d_board2 = newDeviceBoard();
	copyBoardToDevice(origBoard, d_board1);
	dim3 threadDim(getBoardDim(), getBoardDim());
	d_nextNGens<<<1, threadDim>>>(d_board1, d_board2, numGens);
	Board* d_finalBoard_p = pickFinalBoard(&d_board1, &d_board2, numGens);
	copyDeviceToBoard(*d_finalBoard_p, origBoard);
}
