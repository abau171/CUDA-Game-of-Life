#include "hip/hip_runtime.h"
#include <stdio.h>

#include "generation.h"
#include "board.h"
#include "device_board.h"

static __device__ int d_getNumLivingNeighbors(Board board, int x, int y) {
	int numLivingNeighbors = 0;
	for (int dy = -1; dy <= 1; dy++) {
		for (int dx = -1; dx <= 1; dx++) {
			if (dx == 0 && dy == 0) {
				continue;
			}
			if (d_getCellState(board, x + dx, y + dy) == ALIVE) {
				numLivingNeighbors++;
			}
		}
	}
	return numLivingNeighbors;
}

static __device__ CellState getNextState(bool cellAlive, int numLivingNeighbors) {
	CellState nextState = DEAD;
	if (cellAlive) {
		if (numLivingNeighbors >= 2 && numLivingNeighbors <= 3) {
			nextState = ALIVE;
		}
	} else if (numLivingNeighbors == 3) {
		nextState = ALIVE;
	}
	return nextState;
}

__global__ void d_nextNGens(Board d_board1, Board d_board2, int numGens) {
	bool cellAlive = (d_getCellState(d_board1, threadIdx.x, threadIdx.y) == ALIVE);
	int numLivingNeighbors = d_getNumLivingNeighbors(d_board1, threadIdx.x, threadIdx.y);
	CellState nextState = getNextState(cellAlive, numLivingNeighbors);
	d_setCellState(d_board2, threadIdx.x, threadIdx.y, nextState);
}

void nextNGens(Board origBoard, int numGens) {
	Board d_board1 = newDeviceBoard();
	Board d_board2 = newDeviceBoard();
	copyBoardToDevice(origBoard, d_board1);
	dim3 threadDim(BOARD_DIM, BOARD_DIM);
	d_nextNGens<<<1, threadDim>>>(d_board1, d_board2, numGens);
	copyDeviceToBoard(d_board2, origBoard);
}
