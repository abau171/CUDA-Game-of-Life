#include "hip/hip_runtime.h"
#include <stdio.h>

#include "generation.h"
#include "board.h"
#include "device_board.h"

static int getNumLivingNeighbors(Board board, int x, int y) {
	int numLivingNeighbors = 0;
	for (int dy = -1; dy <= 1; dy++) {
		for (int dx = -1; dx <= 1; dx++) {
			if (dx == 0 && dy == 0) {
				continue;
			}
			if (getCellState(board, x + dx, y + dy) == ALIVE) {
				numLivingNeighbors++;
			}
		}
	}
	return numLivingNeighbors;
}

/*static CellState getNextState(Board board, int x, int y) {
	int numLivingNeighbors = getNumLivingNeighbors(board, x, y);
	CellState prevState = getCellState(board, x, y);
	CellState nextState = DEAD;
	if (prevState == ALIVE) {
		if (numLivingNeighbors >= 2 && numLivingNeighbors <= 3) {
			nextState = ALIVE;
		}
	} else if (numLivingNeighbors == 3) {
		nextState = ALIVE;
	}
	return nextState;
}*/

/*static void nextGen(Board prevBoard, Board nextBoard) {
	for (int y = 0; y < BOARD_DIM; y++) {
		for (int x = 0; x < BOARD_DIM; x++) {
			setCellState(nextBoard, x, y, getNextState(prevBoard, x, y));
		}
	}
}*/

__global__ void d_nextNGens(Board d_board, int numGens) {
	for (int i = 0; i < numGens; i++) {
		//nextGen(*curBoard_p, *nextBoard_p);
		//Board* tmp = curBoard_p;
		//curBoard_p = nextBoard_p;
		//nextBoard_p = tmp;
	}
}

void nextNGens(Board origBoard, int numGens) {

	Board d_board = newDeviceBoard();

	//d_nextNGens<<<1, BOAD_SIZE>>>(board, numGens);

	// copy device board back
}
